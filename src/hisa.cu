
#include "hisa.cuh"
#include "io.cuh"
#include "utils.cuh"
#include <cstdint>
#include <iostream>

#include <thrust/adjacent_difference.h>
#include <thrust/binary_search.h>
#include <thrust/device_ptr.h>
#include <thrust/extrema.h>
#include <thrust/fill.h>
#include <thrust/gather.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/iterator/discard_iterator.h>
#include <thrust/iterator/permutation_iterator.h>
#include <thrust/iterator/transform_iterator.h>
#include <thrust/iterator/zip_iterator.h>
#include <thrust/remove.h>
#include <thrust/sequence.h>
#include <thrust/set_operations.h>
#include <thrust/sort.h>
#include <thrust/transform.h>
#include <thrust/unique.h>

namespace vflog {

multi_hisa::multi_hisa(std::string name, int arity, d_buffer_ptr buffer,
                       size_t default_idx) {
    this->arity = arity;
    newt_size = 0;
    full_size = 0;
    delta_size = 0;
    full_columns.resize(arity);
    delta_columns.resize(arity);
    newt_columns.resize(arity);
    data.resize(arity);
    if (buffer) {
        this->buffer = buffer;
    } else {
        this->buffer = std::make_shared<d_buffer>(40960);
    }

    for (int i = 0; i < arity; i++) {
        full_columns[i].column_idx = i;
        delta_columns[i].column_idx = i;
        newt_columns[i].column_idx = i;
        full_columns[i].raw_data = data[i].RAW_PTR;
        delta_columns[i].raw_data = data[i].RAW_PTR;
        newt_columns[i].raw_data = data[i].RAW_PTR;
    }
    set_default_index_column(default_idx);
}

multi_hisa::multi_hisa(std::string name, int arity, const char *filename,
                       d_buffer_ptr buffer, size_t default_idx) {
    name = name;
    this->arity = arity;
    newt_size = 0;
    full_size = 0;
    delta_size = 0;
    full_columns.resize(arity);
    delta_columns.resize(arity);
    newt_columns.resize(arity);
    data.resize(arity);
    name = name;
    if (buffer) {
        this->buffer = buffer;
    } else {
        this->buffer = std::make_shared<d_buffer>(40960);
    }

    for (int i = 0; i < arity; i++) {
        full_columns[i].column_idx = i;
        delta_columns[i].column_idx = i;
        newt_columns[i].column_idx = i;
        full_columns[i].raw_data = data[i].RAW_PTR;
        delta_columns[i].raw_data = data[i].RAW_PTR;
        newt_columns[i].raw_data = data[i].RAW_PTR;
    }
    set_default_index_column(default_idx);

    // read data from file
    read_kary_relation(filename, *this, arity);
    newt_self_deduplicate();
    persist_newt();
}

void multi_hisa::init_load_vectical(
    HOST_VECTOR<HOST_VECTOR<internal_data_type>> &tuples, size_t rows) {
    auto load_start = std::chrono::high_resolution_clock::now();
    auto total_tuples = tuples[0].size();
    for (int i = 0; i < arity; i++) {
        // extract the i-th column
        // thrust::device_vector<internal_data_type> column_data(total_tuples);
        data[i].resize(total_tuples);
        hipMemcpy(data[i].RAW_PTR, tuples[i].data(),
                   tuples[i].size() * sizeof(internal_data_type),
                   hipMemcpyHostToDevice);
        // save columns raw
    }
    this->total_tuples = total_tuples;
    this->newt_size = total_tuples;
    // set newt
    for (int i = 0; i < arity; i++) {
        newt_columns[i].raw_size = total_tuples;
    }
    auto load_end = std::chrono::high_resolution_clock::now();
    this->load_time += std::chrono::duration_cast<std::chrono::microseconds>(
                           load_end - load_start)
                           .count();
}

void multi_hisa::allocate_newt(size_t size) {
    auto old_size = capacity;
    if (total_tuples + size < capacity) {
        // std::cout << "no need to allocate newt" << std::endl;
        // return;
        size = 0;
    }
    // compute offset of each version

    for (int i = 0; i < arity; i++) {
        data[i].resize(old_size + size);
    }
    capacity = old_size + size;

    // newt_size += size;
    // for (int i = 0; i < arity; i++) {
    //     newt_columns[i].raw_data = data[i].data() + old_size;
    // }
}

void multi_hisa::load_column_cpu(VetricalColumnCpu &columns_cpu,
                                 int column_idx) {
    auto total_tuples = columns_cpu.raw_data.size();
    capacity = total_tuples;
    data[column_idx].resize(total_tuples);
    hipMemcpy(data[column_idx].RAW_PTR, columns_cpu.raw_data.data(),
               columns_cpu.raw_data.size() * sizeof(internal_data_type),
               hipMemcpyHostToDevice);
    this->total_tuples = total_tuples;
    this->newt_size = columns_cpu.newt_size;
    this->full_size = columns_cpu.full_size;
    this->delta_size = columns_cpu.delta_size;
    if (columns_cpu.full_size == 0) {
        return;
    }
    // set ptr
    full_columns[column_idx].raw_offset = 0;
    full_columns[column_idx].raw_data = data[column_idx].RAW_PTR;
    delta_columns[column_idx].raw_offset = columns_cpu.delta_head_offset;
    delta_columns[column_idx].raw_data = data[column_idx].RAW_PTR;
    newt_columns[column_idx].raw_offset = columns_cpu.newt_head_offset;
    newt_columns[column_idx].raw_data = data[column_idx].RAW_PTR;
    // copy sorted indices
    if (columns_cpu.full_size != 0) {
        full_columns[column_idx].sorted_indices.resize(columns_cpu.full_size);
        full_columns[column_idx].raw_size = columns_cpu.full_size;
        hipMemcpy(full_columns[column_idx].sorted_indices.RAW_PTR,
                   columns_cpu.full_sorted_indices.data(),
                   columns_cpu.full_sorted_indices.size() *
                       sizeof(internal_data_type),
                   hipMemcpyHostToDevice);
    }
    if (columns_cpu.delta_size != 0) {
        delta_columns[column_idx].sorted_indices.resize(columns_cpu.delta_size);
        delta_columns[column_idx].raw_size = columns_cpu.delta_size;
        hipMemcpy(delta_columns[column_idx].sorted_indices.RAW_PTR,
                   columns_cpu.delta_sorted_indices.data(),
                   columns_cpu.delta_sorted_indices.size() *
                       sizeof(internal_data_type),
                   hipMemcpyHostToDevice);
    }
    if (columns_cpu.newt_size != 0) {
        newt_columns[column_idx].sorted_indices.resize(columns_cpu.newt_size);
        newt_columns[column_idx].raw_size = columns_cpu.newt_size;
        hipMemcpy(newt_columns[column_idx].sorted_indices.RAW_PTR,
                   columns_cpu.newt_sorted_indices.data(),
                   columns_cpu.newt_sorted_indices.size() *
                       sizeof(internal_data_type),
                   hipMemcpyHostToDevice);
    }
}

void multi_hisa::print_all(bool sorted) {
    // print all columns in full
    HOST_VECTOR<internal_data_type> column(total_tuples);
    HOST_VECTOR<internal_data_type> unique_value(total_tuples);
    for (int i = 0; i < arity; i++) {
        thrust::copy(data[i].begin() + full_columns[i].raw_offset,
                     data[i].begin() + full_columns[i].raw_offset +
                         full_columns[i].raw_size,
                     column.begin());
        std::cout << "column data " << i << " " << total_tuples << ":\n";
        for (int j = 0; j < column.size(); j++) {
            std::cout << column[j] << " ";
        }
        std::cout << std::endl;
        std::cout << "unique values " << i << " "
                  << full_columns[i].unique_v.size() << ":\n";
        unique_value.resize(full_columns[i].unique_v.size());
        thrust::copy(full_columns[i].unique_v.begin(),
                     full_columns[i].unique_v.end(), unique_value.begin());
        for (int j = 0; j < full_columns[i].unique_v.size(); j++) {
            std::cout << unique_value[j] << " ";
        }
        std::cout << std::endl;
    }
}

void multi_hisa::print_raw_data(RelationVersion ver) {
    std::printf("print raw data\n");
    HOST_VECTOR<HOST_VECTOR<internal_data_type>> columns_host(arity);
    for (int i = 0; i < arity; i++) {
        columns_host[i].resize(get_versioned_size(ver));
        hipMemcpy(columns_host[i].data(),
                   data[i].RAW_PTR + get_versioned_columns(ver)[i].raw_offset,
                   get_versioned_size(ver) * sizeof(internal_data_type),
                   hipMemcpyDeviceToHost);
    }
    // radix sort host
    thrust::host_vector<internal_data_type> column_host(
        get_versioned_size(ver));
    thrust::host_vector<internal_data_type> sorted_indices_host(
        get_versioned_size(ver));
    thrust::sequence(sorted_indices_host.begin(), sorted_indices_host.end());
    // for (int i = arity - 1; i >= 0; i--) {
    //     auto &column = columns_host[i];
    //     thrust::gather(sorted_indices_host.begin(),
    //     sorted_indices_host.end(),
    //                    column.begin(), column_host.begin());
    //     thrust::stable_sort_by_key(column_host.begin(), column_host.end(),
    //                                sorted_indices_host.begin());
    // }
    // permute the columns
    for (int i = 0; i < arity; i++) {
        thrust::gather(sorted_indices_host.begin(), sorted_indices_host.end(),
                       columns_host[i].begin(), column_host.begin());
        thrust::copy(column_host.begin(), column_host.end(),
                     columns_host[i].begin());
    }

    for (size_t i = 0; i < get_versioned_size(ver); i++) {
        for (int j = 0; j < arity; j++) {
            std::cout << columns_host[j][i] << " ";
        }
        std::cout << std::endl;
    }
}

void multi_hisa::fit() {
    total_tuples = newt_size + full_size;
    for (int i = 0; i < arity; i++) {
        data[i].resize(total_tuples);
        data[i].shrink_to_fit();
    }
}

void multi_hisa::print_stats() {
    std::cout << "sort time: " << sort_time / 1000000.0 << std::endl;
    std::cout << "hash time: " << hash_time / 1000000.0 << std::endl;
    std::cout << "dedup time: " << dedup_time / 1000000.0 << std::endl;
    std::cout << "merge time: " << merge_time / 1000000.0 << std::endl;
}

void multi_hisa::clear() {
    for (int i = 0; i < arity; i++) {
        full_columns[i].raw_offset = 0;
        full_columns[i].sorted_indices.resize(0);
        full_columns[i].sorted_indices.shrink_to_fit();
        full_columns[i].clear_unique_v();

        delta_columns[i].raw_offset = 0;
        delta_columns[i].sorted_indices.resize(0);
        delta_columns[i].sorted_indices.shrink_to_fit();
        delta_columns[i].clear_unique_v();

        newt_columns[i].raw_offset = 0;
        newt_columns[i].sorted_indices.resize(0);
        newt_columns[i].sorted_indices.shrink_to_fit();
        newt_columns[i].clear_unique_v();

        data[i].resize(0);
        data[i].shrink_to_fit();
    }
    newt_size = 0;
    full_size = 0;
    delta_size = 0;
    total_tuples = 0;
}

bool multi_hisa::tuple_exists(std::vector<internal_data_type> &tuple,
                              RelationVersion version) {
    // check if the tuple exists in the relation
    device_data_t tuple_data(arity);

    for (int i = 0; i < arity; i++) {
        tuple_data[i] = tuple[i];
    }

    // create a scalar
    device_data_t input_scalar(1);
    input_scalar[0] = tuple_data[default_index_column];
    device_ranges_t found_range_scalar(1);
    auto &columns = get_versioned_columns(version);
    auto &default_column = columns[default_index_column];
    default_column.map_find(input_scalar.begin(), input_scalar.end(),
                            found_range_scalar.begin());
    device_bitmap_t found_flag(1);

    DEVICE_VECTOR<internal_data_type *> all_col_ptrs(arity);
    for (int i = 0; i < arity; i++) {
        all_col_ptrs[i] = data[i].RAW_PTR + columns[i].raw_offset;
    }
    DEVICE_VECTOR<internal_data_type *> all_idx_ptrs(arity);
    for (int i = 0; i < arity; i++) {
        if (columns[i].sorted_indices.size() != 0) {
            all_idx_ptrs[i] = columns[i].sorted_indices.RAW_PTR;
        } else {
            all_idx_ptrs[i] = nullptr;
        }
    }
    thrust::transform(
        found_range_scalar.begin(), found_range_scalar.end(),
        found_flag.begin(),
        [all_col_ptrs = all_col_ptrs.RAW_PTR,
         all_idx_ptrs = all_idx_ptrs.RAW_PTR, arity = arity,
         default_index_column = default_index_column,
         tuple_data =
             tuple_data.RAW_PTR] __device__(comp_range_t & range) -> bool {
            if (range == UINT32_MAX) {
                return false;
            }
            // get the higher 32 bit of the range, and cast as
            // unsigned int
            auto start = (unsigned int)(range >> 32);
            auto size = (unsigned int)(range & 0xFFFFFFFF);
            for (unsigned int i = 0; i < size; i++) {
                unsigned int exists_tuple_idx =
                    all_idx_ptrs[default_index_column][start + i];
                bool found_flag = true;
                for (int j = 0; j < arity; j++) {
                    if (tuple_data[j] != all_col_ptrs[j][exists_tuple_idx]) {
                        found_flag = false;
                        break;
                    }
                }
                if (found_flag) {
                    return true;
                }
            }
            return false;
        });

    return found_flag[0];
}

ClusteredIndex& multi_hisa::get_clustered_index(RelationVersion version,
                             std::vector<int> column_indices) {
    // get the clustered index of the columns
    if (version == NEWT) {
        for (auto &clustered_index : clustered_indices_newt) {
            if (clustered_index.column_indices == column_indices) {
                return clustered_index;
            }
        }
    } else {
        for (auto &clustered_index : clustered_indices_full) {
            if (clustered_index.column_indices == column_indices) {
                return clustered_index;
            }
        }
    }
    throw std::runtime_error("clustered index not found");
}

} // namespace vflog
